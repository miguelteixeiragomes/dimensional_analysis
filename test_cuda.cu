#include "hip/hip_runtime.h"
#include "test_cuda.cuh"

__global__ void cuda_dims() {
	int32<Time> a(5);
	int32<Time> b(6);
	int64<> c(7);
	int64<> d(8);
	float32<> e(3.14f);
	float64<> f(2.7);
	e = f;
	a + b;
	float64<Length>(8.0) + 7.0_Length;

	{
		+a;
		-a;
		a + b;
		a - b;
		a * b;
		a / b;
		a % b;
		a++;
		a--;
		++a;
		--a;
		(a += b);
		(a -= b);
		(a *= c);
		(a /= d);
		(a == b);
		(a != b);
		(a > b);
		(a < b);
		(a >= b);
		(a <= b);
		(~c);
		(c | d);
		(c ^ d);
		(c & d);
		(c << d);
		(c >> d);
		(c |= d);
		(c &= d);
		(c ^= d);
		(c <<= d);
		(c >>= d);
	}
	{
		(c + 7);
		(7 - d);
		(c * 7);
		(7 / d);
		(c == 7);
		(7 != c);
		(c > 7);
		(7 < d);
		(c >= 7);
		(7 <= d);
		(c | 7);
		(7 ^ d);
		(c & 7);
		(7 << d);
		(c >> 7);
	}

	
	float32<Time> z = powf<7>(powf<1, 7>(float32<Time>(7.2487)));
	float32<Length>(5) + sqrtf(float32<Area>(25));
	float64<Length>(5) + cbrt(float64<Volume>(125));
	
	float64<Velocity> lightspeed = 1.0 / sqrt(float64<Permittivity>(8.85418781762039e-12) * float64<Permeability>(1.256637061435917e-6));
}

void test_cuda() {
	cuda_dims<<<2, 2>>>();
}