#include "hip/hip_runtime.h"
#include "test_cuda.cuh"

__global__ void cuda_dims() {
	int32<Time> a(5);
	int32<Time> b(6);
	int64<> c(7);
	int64<> d(8);
	float32<> e(3.14f);
	float64<> f(2.7);
	e = f;
	a + b;
	
	pow<7>(pow<1, 7>(float32<Time>(7.2487)));
	float32<Length>(5) + sqrtf(float32<Area>(25));
	float64<Length>(5) + cbrt(float64<Volume>(125));
	
	float64<Velocity> lightspeed = 1.0 / sqrt(float64<Permittivity>(8.85418781762039e-12) * float64<Permeability>(1.256637061435917e-6));
}

void test_cuda() {
	cuda_dims<<<2, 2>>>();
}